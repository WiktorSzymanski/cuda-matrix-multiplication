#include "hip/hip_runtime.h"

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>

__global__ void matrixMul(int *a, int *b, int *c, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int index = row * N + col;

  if (row < N && col < N) {
    int sum = 0;
    for (int i = 0; i < N; i++) {
      sum += a[row * N + i] * b[i * N + col];
    }
    c[index] = sum;
  }
}

void cpuMatrixMul(std::vector<int> &a, std::vector<int> &b, std::vector<int> &c, int N) {
  int tmp;
  for (int row = 0; row < N; row++) {
    for (int col = 0; col < N; col++) {
      tmp = 0;
      for (int i = 0; i < N; i++) {
        tmp += a[row * N + i] * b[i * N + col];
      }
      c[row * N + col] = tmp;
    }
  }
}

int main(int argc, char **argv) {
  if (argc != 2) {
    std::cout << "Usage: " << argv[0] << " <N>" << std::endl;
    return 1;
  }
  int N = atoi(argv[1]);
  size_t size = N * N * sizeof(int);

  std::vector<int> a(N * N);
  std::vector<int> b(N * N);
  std::vector<int> c(N * N);
  std::vector<int> cCPU(N * N);

  std::generate(a.begin(), a.end(), []() { return rand() % 100; });
  std::generate(b.begin(), b.end(), []() { return rand() % 100; });

  int *cA, *cB, *cC;
  hipMalloc(&cA, size);
  hipMalloc(&cB, size);
  hipMalloc(&cC, size);

  hipMemcpy(cA, a.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(cB, b.data(), size, hipMemcpyHostToDevice);

  int threadsPerBlock = 16;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  dim3 threads(threadsPerBlock, threadsPerBlock);
  dim3 blocks(blocksPerGrid, blocksPerGrid);

  auto start = std::chrono::high_resolution_clock::now();
  matrixMul<<<blocks, threads>>>(cA, cB, cC, N);
  hipMemcpy(c.data(), cC, size, hipMemcpyDeviceToHost);
  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "GPU time: " << duration.count() / 1000.0 << " ms" << std::endl;

  start = std::chrono::high_resolution_clock::now();
  cpuMatrixMul(a, b, cCPU, N);
  end = std::chrono::high_resolution_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "CPU time: " << duration.count() / 1000.0 << " ms" << std::endl;

  if (c == cCPU) {
    printf("Success!\n");
  } else {
    printf("Failed!\n");
  }

  return 0;
}